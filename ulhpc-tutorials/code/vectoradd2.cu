#include <cstdio>
#include "hip/hip_runtime.h"

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i) {
    a[i] = num;
  }
}

/*
 * Device kernel to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

__global__
void initWithKernel(float num, float *a, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride) {
    a[i] = num;
  }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride) {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++) {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a, *b, *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 512;
  numberOfBlocks = 80;

  initWithKernel<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWithKernel<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWithKernel<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  hipDeviceSynchronize();

  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}

