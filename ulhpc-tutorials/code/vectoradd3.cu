#include "hip/hip_runtime.h"
#include <cstdio>
#include "hip/hip_runtime.h"

/*
 * Host function to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i) {
    a[i] = num;
  }
}

/*
 * Device kernel to initialize vector elements. This function
 * simply initializes each element to equal its index in the
 * vector.
 */

__global__
void initWithKernel(float num, float *a, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride) {
    a[i] = num;
  }
}

/*
 * Device kernel stores into `result` the sum of each
 * same-indexed value of `a` and `b`.
 */

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride) {
    result[i] = a[i] + b[i];
  }
}

/*
 * Host function to confirm values in `vector`. This function
 * assumes all values are the same `target` value.
 */

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++) {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  hipGetDevice(&deviceId);
  printf("Device Id %d\n", deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a, *b, *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 512;
  numberOfBlocks = 80;

  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);

  initWithKernel<<<numberOfBlocks, threadsPerBlock>>>(3, a, N);
  initWithKernel<<<numberOfBlocks, threadsPerBlock>>>(4, b, N);
  initWithKernel<<<numberOfBlocks, threadsPerBlock>>>(0, c, N);

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  hipMemPrefetchAsync(c, size, hipCpuDeviceId);
  
  hipDeviceSynchronize();
  
  checkElementsAre(7, c, N);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}

