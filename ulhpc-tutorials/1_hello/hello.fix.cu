#include <iostream>
#include <cstdio>
#include "hip/hip_runtime.h"

using namespace std;

void helloCPU()
{
  std::cout<<"Hello from Cpu.\n";
}

__global__
void helloGPU()
{
  printf("Hello also from Gpu.\n");
}

int main()
{

  helloCPU();
  helloGPU<<<5, 5>>>();

  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}
