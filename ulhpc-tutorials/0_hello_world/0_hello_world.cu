#include <cstdio>
#include "hip/hip_runtime.h"

void CPUFunction()
{
  printf("hello from the Cpu.\n");
}

__global__
void GPUFunction()
{
  printf("hello from the Gpu.\n");
}

int main()
{
  CPUFunction();

  GPUFunction<<<1, 1>>>();

  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}
