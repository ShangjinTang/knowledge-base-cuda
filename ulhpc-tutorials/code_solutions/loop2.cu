#include <cstdio>
#include "hip/hip_runtime.h"

__global__
void loop()
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("This is iteration number %d\n", i);
}

int main()
{
  /*
   * When refactoring 'loop' to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use more than 1 block in
   * the execution configuration.
   */

  int block_size = 2;
  int kernel_size = 5;
  loop<<<block_size, kernel_size>>>();

  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}
