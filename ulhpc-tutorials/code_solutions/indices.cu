#include <cstdio>
#include "hip/hip_runtime.h"

__global__
void printif()
{
  if (threadIdx.x == 1023 && blockIdx.x == 255) {
    printf("Success!\n");
  }
}

int main()
{
  /*
   * Update the execution configuration so that the kernel
   * will print `"Success!"`.
   */

  printif<<<256, 1024>>>();

  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}
