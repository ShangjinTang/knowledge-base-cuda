#include <cstdio>
#include "hip/hip_runtime.h"

__global__
void loop()
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("This is iteration number %d\n", i);
}

int main()
{
  /*
   * When refactoring 'loop' to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * Use 1 block of threads.
   */

  int block_size = 1;
  int kernel_size = 10;
  loop<<<block_size, kernel_size>>>();

  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}
